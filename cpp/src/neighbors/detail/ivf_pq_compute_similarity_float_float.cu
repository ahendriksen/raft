
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/neighbors/detail/ivf_pq_compute_similarity-inl.cuh>
#include <raft/neighbors/detail/ivf_pq_fp_8bit.cuh>

#define instantiate_raft_neighbors_ivf_pq_detail_compute_similarity_select(OutT, LutT)  \
  template auto raft::neighbors::ivf_pq::detail::compute_similarity_select<OutT, LutT>( \
    const hipDeviceProp_t& dev_props,                                                    \
    bool manage_local_topk,                                                             \
    int locality_hint,                                                                  \
    double preferred_shmem_carveout,                                                    \
    uint32_t pq_bits,                                                                   \
    uint32_t pq_dim,                                                                    \
    uint32_t precomp_data_count,                                                        \
    uint32_t n_queries,                                                                 \
    uint32_t n_probes,                                                                  \
    uint32_t topk)                                                                      \
    ->raft::neighbors::ivf_pq::detail::selected<OutT, LutT>;

#define COMMA ,
instantiate_raft_neighbors_ivf_pq_detail_compute_similarity_select(float, float);

#undef COMMA

#undef instantiate_raft_neighbors_ivf_pq_detail_compute_similarity_select
